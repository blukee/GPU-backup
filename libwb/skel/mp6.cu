#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

#define Mask_width  5
#define Mask_radius Mask_width/2
#define TILE_WIDTH 16
#define CHANNEL 3
#define O_TILE_WIDTH 12
#define BLOCK_WIDTH O_TILE_WIDTH + (Mask_width-1)
#define CLAMP(X) min(max(X,0.0f),1.0f)

//@@ INSERT CODE HERE
__global__ void imageConvolution3D(float * I, float * O, const float * __restrict__  M,
                   int IWidth, int IHeight, int IChannel,
                   int numMRows, int numMColumns) {
	//const int TILE_WIDTH = 16;
	//const int CHANNEL = 3;
    //@@ Insert code to implement matrix multiplication here
	__shared__ float ds_I[TILE_WIDTH][TILE_WIDTH][CHANNEL];
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	
	int row_o = blockIdx.y*O_TILE_WIDTH + ty;
	int col_o = blockIdx.x*O_TILE_WIDTH + tx;
	int row_i = row_o - Mask_radius;
	int col_i = col_o - Mask_radius; 
	
	// load image data into shared memory
	if((row_i >= 0) && (row_i < IHeight) && 
		(col_i >= 0) && (col_i < IWidth) ) {
		ds_I[ty][tx][tz] = I[(row_i*IWidth + col_i) * IChannel + tz];
	} else{
		ds_I[ty][tx][tz] = 0.0f;
	}
	__syncthreads();
	
	float accum = 0.0;
	if(ty < O_TILE_WIDTH && tx < O_TILE_WIDTH){
		for(int i = 0; i < Mask_width; i++) {
			for(int j  = 0; j < Mask_width; j++) {
				accum += M[i*numMRows + j] * ds_I[i+ty][j+tx][tz];
			}
		}
	}

	if(row_o < IHeight && col_o < IWidth)
		O[(row_o*IWidth + col_o) * IChannel + tz] = CLAMP(accum);
}

//2D convolution, load memory 
__global__ void imageConvolution2D(float * I, float * O, const float * __restrict__  M,
                   int IWidth, int IHeight, int IChannel,
                   int numMRows, int numMColumns) {
    //@@ Insert code to implement matrix multiplication here
	__shared__ float ds_I[TILE_WIDTH][TILE_WIDTH][CHANNEL];
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int row_o = blockIdx.y*O_TILE_WIDTH + ty;
	int col_o = blockIdx.x*O_TILE_WIDTH + tx;
	int row_i = row_o - Mask_radius;
	int col_i = col_o - Mask_radius; 
	
	// load image data into shared memory
	if((row_i >= 0) && (row_i < IHeight) && 
		(col_i >= 0) && (col_i < IWidth) ) {
		int idxI = (row_i*IWidth + col_i) * IChannel;
		ds_I[ty][tx][0] = I[idxI];
		ds_I[ty][tx][1] = I[idxI + 1];
		ds_I[ty][tx][2] = I[idxI + 2];
	} else{
		ds_I[ty][tx][0] = 0.0f;
		ds_I[ty][tx][1] = 0.0f;
		ds_I[ty][tx][2] = 0.0f;
	}
	
	__syncthreads(); 
	

	for(int k = 0; k < IChannel; k++) {
		float accum0 = 0.0;
		if(ty < O_TILE_WIDTH && tx < O_TILE_WIDTH){
			for(int i = 0; i < Mask_width; i++) {
				for(int j  = 0; j < Mask_width; j++) {
					accum0 += M[i*Mask_width + j] * ds_I[i+ty][j+tx][k];
				}
			}
		}
	
		if(row_o < IHeight && col_o < IWidth) {
			O[(row_o*IWidth + col_o) * IChannel + k] = CLAMP(accum0);
		}
		
		if(row_o == 0 && col_o == 24) {
			O[(row_o*IWidth + col_o) * IChannel + k] = 0.55;
		}
	}
	__syncthreads();
}

__global__
void convolution2D (float * I, const float * __restrict__ M, float * P,
        int channels, int width, int height)
{
    __shared__ float N_ds[O_TILE_WIDTH][O_TILE_WIDTH];

    int bx = blockIdx.x,  by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    for (int k = 0; k < channels; ++k) {
        int dest  = ty * TILE_WIDTH + tx;
        int destX = dest % O_TILE_WIDTH;
        int destY = dest / O_TILE_WIDTH;
        int srcY  = by * TILE_WIDTH + destY - Mask_radius;
        int srcX  = bx * TILE_WIDTH + destX - Mask_radius;
        int src   = (srcY * width + srcX) * channels + k;

        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
        else
            N_ds[destY][destX] = 0.0;

        dest  = ty * TILE_WIDTH + tx + TILE_WIDTH * TILE_WIDTH;
        destY = dest / O_TILE_WIDTH;
        destX = dest % O_TILE_WIDTH;
        srcY  = by * TILE_WIDTH + destY - Mask_radius;
        srcX  = bx * TILE_WIDTH + destX - Mask_radius;
        src   = (srcY * width + srcX) * channels + k;

        if (destY < O_TILE_WIDTH) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = I[src];
            else
                N_ds[destY][destX] = 0.0;
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < Mask_width; ++y)
            for (int x = 0; x < Mask_width; ++x)
                accum += N_ds[ty + y][tx + x] * M[y * Mask_width + x];

        int y = by * TILE_WIDTH + ty;
        int x = bx * TILE_WIDTH + tx;
        if (y < height && x < width)
            P[(y * width + x) * channels + k] = min(max(accum, 0.0), 1.0);

        __syncthreads();
    }
}

int main(int argc, char* argv[]) {
    wbArg_t args;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);
    inputMaskFile = wbArg_getInputFile(args, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

	wbLog(TRACE, "The dimensions of image are ", imageWidth, " x ", imageHeight, " x " ,imageChannels);
	
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");


    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
	//@@ Initialize the grid and block dimensions here
    dim3 DimGrid((imageWidth-1)/O_TILE_WIDTH + 1, (imageHeight-1)/O_TILE_WIDTH + 1, 1);
	dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);			 
	
    //@@ Launch the GPU Kernel here
	imageConvolution2D<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, deviceMaskData, imageWidth, imageHeight, imageChannels, maskRows, maskColumns);
	//convolution2D<<<DimGrid, DimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData, imageChannels, imageWidth, imageHeight);
//    hipDeviceSynchronize();
    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(args, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
